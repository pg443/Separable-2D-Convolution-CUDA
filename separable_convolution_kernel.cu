
#include <hip/hip_runtime.h>
/* FIXME: Edit this file to complete the functionality of 2D separable 
 * convolution on the GPU. You may add additional kernel functions 
 * as necessary. 
 */

__global__ void convolve_rows_kernel_naive(float *result, float *input, float *kernel,\
                                            int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;
    y = threadIdx.x + blockIdx.x * blockDim.x;

    for (x = 0; x < num_cols; x++) {
        j1 = x - half_width;
        j2 = x + half_width;
        /* Clamp at the edges of the matrix */
        if (j1 < 0) 
            j1 = 0;
        if (j2 >= num_cols) 
            j2 = num_cols - 1;

        /* Obtain relative position of starting element from element being convolved */
        i1 = j1 - x; 
        
        j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
        j2 = j2 - x + half_width;

        /* Convolve along row */
        result[y * num_cols + x] = 0.0f;
        for(i = i1, j = j1; j <= j2; j++, i++)
            result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + i];
    }
}

__global__ void convolve_columns_kernel_naive(float *result, float *input, float *kernel,\
                                                int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    y = threadIdx.x + blockIdx.x * blockDim.x;

    for(x = 0; x < num_cols; x++) {
        j1 = y - half_width;
        j2 = y + half_width;
        /* Clamp at the edges of the matrix */
        if (j1 < 0) 
            j1 = 0;
        if (j2 >= num_rows) 
            j2 = num_rows - 1;

        /* Obtain relative position of starting element from element being convolved */
        i1 = j1 - y; 
        
        j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
        j2 = j2 - y + half_width;

        /* Convolve along column */            
        result[y * num_cols + x] = 0.0f;
        for (i = i1, j = j1; j <= j2; j++, i++)
            result[y * num_cols + x] += 
                kernel[j] * input[y * num_cols + x + (i * num_cols)];
    }
}

__global__ void convolve_rows_kernel_optimized()
{
    return;
}

__global__ void convolve_columns_kernel_optimized()
{
    return;
}





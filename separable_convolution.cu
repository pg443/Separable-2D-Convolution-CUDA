#include "hip/hip_runtime.h"
/* Host code that implements a  separable convolution filter of a 
 * 2D signal with a gaussian kernel.
 * 
 * Author: Naga Kandasamy
 * Date modified: May 26, 2020
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

extern "C" void compute_gold(float *, float *, int, int, int);
extern "C" float *create_kernel(float, int);
void print_kernel(float *, int);
void print_matrix(float *, int, int);
void compute_on_device(float *, float *, float *, int, int, int);
void check_for_error(const char *);

/* Width of convolution kernel */
#define HALF_WIDTH 8
#define COEFF 10

__constant__ float kernel_c[2 * HALF_WIDTH + 1]; /* Allocation for the kernel in GPU constant memory */


/* Uncomment line below to spit out debug information */
// #define DEBUG

#define THREAD_BLOCK 256

/* Include device code */
#include "separable_convolution_kernel.cu"

int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("Usage: %s num-rows num-columns\n", argv[0]);
        printf("num-rows: height of the matrix\n");
        printf("num-columns: width of the matrix\n");
        exit(EXIT_FAILURE);
    }

    int num_rows = atoi(argv[1]);
    int num_cols = atoi(argv[2]);

    /* Create input matrix */
    int num_elements = num_rows * num_cols;
    printf("Creating input matrix of %d x %d\n", num_rows, num_cols);
    float *matrix_a = (float *)malloc(sizeof(float) * num_elements);
    float *matrix_c = (float *)malloc(sizeof(float) * num_elements);
	
    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++) {
        matrix_a[i] = rand()/(float)RAND_MAX;			 
        matrix_c[i] = matrix_a[i]; /* Copy contents of matrix_a into matrix_c */
    }
	 
	/* Create Gaussian kernel */	  
    float *gaussian_kernel = create_kernel((float)COEFF, HALF_WIDTH);	
#ifdef DEBUG
    print_kernel(gaussian_kernel, HALF_WIDTH); 
#endif  

    /* Convolve matrix along rows and columns. 
       The result is stored in matrix_a, thereby overwriting the 
       original contents of matrix_a.		
     */
    printf("\nConvolving the matrix on the CPU\n");	  
    struct timeval start, stop;	
    gettimeofday(&start, NULL);

    compute_gold(matrix_a, gaussian_kernel, num_cols,\
                  num_rows, HALF_WIDTH);
    
    gettimeofday(&stop, NULL);
    printf("Execution time for the CPU= %f \n",stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000);
    
#ifdef DEBUG	 
    print_matrix(matrix_a, num_cols, num_rows);
#endif
  
    float *gpu_result = (float *)malloc(sizeof(float) * num_elements);
    
    /* FIXME: Edit this function to complete the functionality on the GPU.
       The input matrix is matrix_c and the result must be stored in 
       gpu_result.
     */
    printf("\nConvolving matrix on the GPU\n");
    compute_on_device(gpu_result, matrix_c, gaussian_kernel, num_cols,\
                       num_rows, HALF_WIDTH);
       
    printf("\nComparing CPU and GPU results\n");
    float sum_delta = 0, sum_ref = 0;
    for (i = 0; i < num_elements; i++) {
        sum_delta += fabsf(matrix_a[i] - gpu_result[i]);
        sum_ref   += fabsf(matrix_a[i]);
    }
        
    float L1norm = sum_delta / sum_ref;
    float eps = 1e-6;
    printf("L1 norm: %E\n", L1norm);
    printf((L1norm < eps) ? "TEST PASSED\n" : "TEST FAILED\n");

    free(matrix_a);
    free(matrix_c);
    free(gpu_result);
    free(gaussian_kernel);

    exit(EXIT_SUCCESS);
}
/* FIXME: Edit this function to compute the convolution on the device.*/
void compute_on_device(float *gpu_result, float *matrix_c, float *gaussian_kernel,\
                            int num_cols, int num_rows, int half_width)
{
    float *matrix_d = NULL;
    hipMalloc((void**)&matrix_d, num_cols * num_rows * sizeof(float));
    hipMemcpy(matrix_d, matrix_c, num_cols * num_rows * sizeof(float), hipMemcpyHostToDevice);
    
    float *gaussian_kernel_d = NULL;
    hipMalloc((void**)&gaussian_kernel_d, (2 * half_width + 1) * sizeof(float));
    hipMemcpy(gaussian_kernel_d, gaussian_kernel, (2 * half_width + 1) * sizeof(float), hipMemcpyHostToDevice);
    

    float *matrix_temp_d = NULL;
    hipMalloc((void**)&matrix_temp_d, num_cols * num_rows * sizeof(float));
    
    float *gpu_result_d = NULL;
    hipMalloc((void**)&gpu_result_d, num_cols * num_rows * sizeof(float));

    
    dim3 thread_block(THREAD_BLOCK, 1, 1);
    int num_block = (num_rows + THREAD_BLOCK - 1) / THREAD_BLOCK;
    dim3 grid(num_block, 1); 

    struct timeval start, stop;	
    gettimeofday(&start, NULL);
    
    printf("Using global memory for convolution\n");    
	convolve_rows_kernel_naive<<<grid, thread_block>>>(matrix_temp_d,  matrix_d, gaussian_kernel_d,\
                                                         num_cols,  num_rows,  half_width); 
    hipDeviceSynchronize();
	check_for_error("KERNEL FAILURE ROW");
    
	convolve_columns_kernel_naive<<<grid, thread_block>>>(gpu_result_d, matrix_temp_d,  gaussian_kernel_d,\
                                                             num_cols,  num_rows,  half_width); 
    hipDeviceSynchronize();
    gettimeofday(&stop, NULL);
	printf("Execution time for the GPU= %f \n",stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000);
    check_for_error("KERNEL FAILURE COLUMN");
    
    hipMemcpy(gpu_result, gpu_result_d, num_rows * num_cols * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(matrix_d);
    hipFree(matrix_temp_d);
    hipFree(gpu_result_d);
    hipFree(gaussian_kernel_d);
}

/* Check for errors reported by the CUDA run time */
void check_for_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA ERROR: %s (%s)\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    return;
} 

/* Print convolution kernel */
void print_kernel(float *kernel, int half_width)
{
    int i, j = 0;
    for (i = -half_width; i <= half_width; i++) {
        printf("%0.2f ", kernel[j]);
        j++;
    }

    printf("\n");
    return;
}

/* Print matrix */
void print_matrix(float *matrix, int num_cols, int num_rows)
{
    int i,  j;
    float element;
    for (i = 0; i < num_rows; i++) {
        for (j = 0; j < num_cols; j++){
            element = matrix[i * num_cols + j];
            printf("%0.2f ", element);
        }
        printf("\n");
    }

    return;
}

